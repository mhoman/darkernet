#include "hip/hip_runtime.h"
#include "cuda_runtime"
#include "hiprand.h"
#include "cublas_v2"

extern "C" {
#include "region_layer.h"
}

__device__ void entry_index_gpu() {
  // entry_index(layer l, int batch, int location, int entry)
  // int n =   location / (l.w*l.h);
  // int loc = location % (l.w*l.h);
  // return n * l.w * l.h * (l.coords+l.classes+1) + entry * l.w * l.h + loc;
}

__global__ void region_boxes_kernel(layer l, int w, int h,
  int netw, int neth, float thresh, float **probs, box *boxes,
  int only_objectness, int *map, float tree_thresh, int relative,
  int wh) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // int n =

  // unsigned int ix =
  // unsigned int iy =
  // unsigned int iz =
}

void forward_getregion_boxes_gpu(layer l, int w, int h,
  int netw, int neth, float thresh, float **probs, box *boxes,
  int only_objectness, int *map, float tree_thresh, int relative) {

  dim3 block(512);
  int size = (l.batch * l.w * l.h * l.n / block.x - 1) / block.x
  dim3 grid(size);
  int wh = w * h;

  region_boxes_kernel<<< grid, block >>>(l, w, h,
    netw, neth, thresh, probs, box boxes,
    only_objectness, map, tree_thresh, relative,
    wh);
}

//
// void my_forward_region_layer_gpu(const layer l, network net)
// {
//     int i,j,b,t,n;
//     memcpy(l.output, net.input, l.outputs*l.batch*sizeof(float));
//
// #ifndef GPU
//     for (b = 0; b < l.batch; ++b){
//         for(n = 0; n < l.n; ++n){
//             int index = entry_index(l, b, n*l.w*l.h, 0);
//             activate_array(l.output + index, 2*l.w*l.h, LOGISTIC);
//             index = entry_index(l, b, n*l.w*l.h, 4);
//             if(!l.background) activate_array(l.output + index,   l.w*l.h, LOGISTIC);
//         }
//     }
//     if (l.softmax){
//         int index = entry_index(l, 0, 0, l.coords + !l.background);
//         softmax_cpu(net.input + index, l.classes + l.background, l.batch*l.n, l.inputs/l.n, l.w*l.h, 1, l.w*l.h, 1, l.output + index);
//     }
// #endif
//
//     memset(l.delta, 0, l.outputs * l.batch * sizeof(float));
//     if(!net.train) return;
//     float avg_iou = 0;
//     float recall = 0;
//     float avg_cat = 0;
//     float avg_obj = 0;
//     float avg_anyobj = 0;
//     int count = 0;
//     int class_count = 0;
//     *(l.cost) = 0;
//     for (b = 0; b < l.batch; ++b) {
//         for (j = 0; j < l.h; ++j) {
//             for (i = 0; i < l.w; ++i) {
//                 for (n = 0; n < l.n; ++n) {
//                     int box_index = entry_index(l, b, n*l.w*l.h + j*l.w + i, 0);
//                     box pred = get_region_box(l.output, l.biases, n, box_index, i, j, l.w, l.h, l.w*l.h);
//                     float best_iou = 0;
//                     for(t = 0; t < 30; ++t){
//                         box truth = float_to_box(net.truth + t*5 + b*l.truths, 1);
//                         if(!truth.x) break;
//                         float iou = box_iou(pred, truth);
//                         if (iou > best_iou) {
//                             best_iou = iou;
//                         }
//                     }
//                     int obj_index = entry_index(l, b, n*l.w*l.h + j*l.w + i, 4);
//                     avg_anyobj += l.output[obj_index];
//                     l.delta[obj_index] = l.noobject_scale * (0 - l.output[obj_index]);
//                     if(l.background) l.delta[obj_index] = l.noobject_scale * (1 - l.output[obj_index]);
//                     if (best_iou > l.thresh) {
//                         l.delta[obj_index] = 0;
//                     }
//
//                     if(*(net.seen) < 12800){
//                         box truth = {0};
//                         truth.x = (i + .5)/l.w;
//                         truth.y = (j + .5)/l.h;
//                         truth.w = l.biases[2*n]/l.w;
//                         truth.h = l.biases[2*n+1]/l.h;
//                         delta_region_box(truth, l.output, l.biases, n, box_index, i, j, l.w, l.h, l.delta, .01, l.w*l.h);
//                     }
//                 }
//             }
//         }
//         for(t = 0; t < 30; ++t){
//             box truth = float_to_box(net.truth + t*5 + b*l.truths, 1);
//
//             if(!truth.x) break;
//             float best_iou = 0;
//             int best_n = 0;
//             i = (truth.x * l.w);
//             j = (truth.y * l.h);
//             //printf("%d %f %d %f\n", i, truth.x*l.w, j, truth.y*l.h);
//             box truth_shift = truth;
//             truth_shift.x = 0;
//             truth_shift.y = 0;
//             //printf("index %d %d\n",i, j);
//             for(n = 0; n < l.n; ++n){
//                 int box_index = entry_index(l, b, n*l.w*l.h + j*l.w + i, 0);
//                 box pred = get_region_box(l.output, l.biases, n, box_index, i, j, l.w, l.h, l.w*l.h);
//                 if(l.bias_match){
//                     pred.w = l.biases[2*n]/l.w;
//                     pred.h = l.biases[2*n+1]/l.h;
//                 }
//                 //printf("pred: (%f, %f) %f x %f\n", pred.x, pred.y, pred.w, pred.h);
//                 pred.x = 0;
//                 pred.y = 0;
//                 float iou = box_iou(pred, truth_shift);
//                 if (iou > best_iou){
//                     best_iou = iou;
//                     best_n = n;
//                 }
//             }
//             //printf("%d %f (%f, %f) %f x %f\n", best_n, best_iou, truth.x, truth.y, truth.w, truth.h);
//
//             int box_index = entry_index(l, b, best_n*l.w*l.h + j*l.w + i, 0);
//             float iou = delta_region_box(truth, l.output, l.biases, best_n, box_index, i, j, l.w, l.h, l.delta, l.coord_scale *  (2 - truth.w*truth.h), l.w*l.h);
//             if(iou > .5) recall += 1;
//             avg_iou += iou;
//
//             //l.delta[best_index + 4] = iou - l.output[best_index + 4];
//             int obj_index = entry_index(l, b, best_n*l.w*l.h + j*l.w + i, l.coords);
//             avg_obj += l.output[obj_index];
//             l.delta[obj_index] = l.object_scale * (1 - l.output[obj_index]);
//             if (l.rescore) {
//                 l.delta[obj_index] = l.object_scale * (iou - l.output[obj_index]);
//             }
//             if(l.background){
//                 l.delta[obj_index] = l.object_scale * (0 - l.output[obj_index]);
//             }
//
//             int class = net.truth[t*(l.coords + 1) + b*l.truths + l.coords];
//             if (l.map) class = l.map[class];
//             int class_index = entry_index(l, b, best_n*l.w*l.h + j*l.w + i, l.coords + 1);
//             delta_region_class(l.output, l.delta, class_index, class, l.classes, l.softmax_tree, l.class_scale, l.w*l.h, &avg_cat);
//             ++count;
//             ++class_count;
//         }
//     }
//     //printf("\n");
//     *(l.cost) = pow(mag_array(l.delta, l.outputs * l.batch), 2);
//     printf("Region Avg IOU: %f, Class: %f, Obj: %f, No Obj: %f, Avg Recall: %f,  count: %d\n", avg_iou/count, avg_cat/class_count, avg_obj/count, avg_anyobj/(l.w*l.h*l.n*l.batch), recall/count, count);
// }
